#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <chrono>

#include "tbb/tbb.h"

#include <hipcub/hipcub.hpp>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <thread>

#include "crystal/crystal.cuh"

using namespace hipcub;
using namespace std;
using namespace tbb;

void runCPU(int* values, int size, int offset) {

    parallel_for( blocked_range<int>(offset, offset+size),
                       [&](blocked_range<int> r)
    {
    	// int worker_index = tbb::task_arena::current_thread_index();
    	//printf("worker_index = %d\n", worker_index);
        for (int i=r.begin(); i<r.end(); ++i)
        {
            values[i] = values[i] * values[i];
            //printf("index = %d\n", i);
        }
    });
}

__global__ void kernel2D(int** d_A, int row, int cols) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid < cols) {
		d_A[row][tid] = d_A[row][tid] * d_A[row][tid];
	}
}

void runGPU2D(int** d_A, int row, int cols) {

    hipStream_t stream;
    hipStreamCreate(&stream);

    kernel2D<<<(cols + 128 - 1)/128, 128, 0, stream>>>(d_A, row, cols); 

    hipStreamDestroy(stream);
}

__global__ void kernel1D(int* d_A, int row, int cols) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid < cols) {
		d_A[row * cols + tid] = d_A[row * cols + tid] * d_A[row * cols + tid];
	}
}

void runGPU1D(int* d_A, int row, int cols) {

    hipStream_t stream;
    hipStreamCreate(&stream);

    kernel1D<<<(cols + 128 - 1)/128, 128, 0, stream>>>(d_A, row, cols); 

    hipStreamDestroy(stream);
}

int main() {

	int rows = 64;
	int cols = 1024 * 1024;

	int** A = new int*[rows];
	A[0] = new int[rows * cols];
	for (int i = 1; i < rows; ++i) A[i] = A[i-1] + cols;

	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {
			A[i][j] = i*cols+j;
		}
	}

	int** temp = new int*[rows];
	hipMalloc((void**) &(temp[0]), rows * cols * sizeof(int));
	for (int i = 1; i < rows; ++i) temp[i] = temp[i-1] + cols;
	hipMemcpy(temp[0], A[0], rows * cols * sizeof(int), hipMemcpyHostToDevice);

	int** d_A;
	hipMalloc((void**) &(d_A), rows * sizeof(int*));
	hipMemcpy(d_A, temp, rows * sizeof(int*), hipMemcpyHostToDevice);


	hipEvent_t start, stop; 
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop); 
	hipEventRecord(start, 0);

	parallel_for(int(0), rows, [=](int i){
		runGPU2D(d_A, i, cols);
	});

	// parallel_for(int(0), rows, [=](int i){
	// 	runGPU1D(temp[0], i, cols);
	// });

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	cout << "Time Taken Total: " << time << endl;


	int** B = new int*[rows];
	B[0] = new int[rows * cols];	
	for (int i = 1; i < rows; ++i) B[i] = B[i-1] + cols;
	hipMemcpy(B[0], temp[0], rows * cols * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < rows; ++i) {
		for (int j = 0; j < cols; ++j) {
			assert(B[i][j] == A[i][j] * A[i][j]);
		}
	}	

	return 0;

}